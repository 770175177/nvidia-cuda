
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void myKernel() 
{
    printf("grid(%d/%d,%d/%d)block(%d/%d, %d/%d) Hello, world from the device!\n",
		blockIdx.x, gridDim.x, blockIdx.y, gridDim.y,
	       	threadIdx.x, blockDim.x, threadIdx.y, blockDim.y); 
} 

int main() 
{ 
    dim3 grid(3, 2);
    dim3 block(2, 2);

    myKernel<<<grid, block>>>();
    hipError_t hipError_t = hipGetLastError();

    if(hipError_t != hipSuccess) {
        printf("cuda execute error: %s\n", hipGetErrorString(hipError_t));
        return -1;
    } else {
        printf("cuda execute success\n");
    }

    hipDeviceSynchronize();

    return 0;
} 

